#include <iostream>
#include "NWGlobal.h"
#include "NWAnalysis_GPU.cuh"
#include "NWSimParameters.h"
#include "hip/hip_runtime.h"
#include ""
#include <sstream>

void AnalysisTheResult(int argc, char* argv[]) {
	/*Local Vars*/
	stringstream ss;
	int deviceCount;
	int currentDevice;
	int selectedDeviceStart;
	int numberOfSelectedDevice;
	hipError_t theError;
	double ParticleEnergy;
	/*Body*/

	hipGetDeviceCount(&deviceCount);

	hipGetDevice(&currentDevice);

	NWAnalysis_GPU analysis;

	if (argc < 4) {
		std::cout << "The argument number is: " << argc << std::endl;
		std::cout << "You must special the 1) the original file path" << std::endl;
		std::cout << "You must special the 2) the start of selectted GPU ID" << std::endl;
		std::cout << "You must special the 3) the number of selectted GPU ID" << std::endl;
		std::cout << "You must special the 4) the particle energy" << std::endl;
		getchar();
		exit(1);
	}
	std::string originalDataFilePath;

	originalDataFilePath = std::string(argv[1]);

	ss.clear();
	ss.str("");
	ss << argv[2];
	ss >> selectedDeviceStart;

	ss.clear();
	ss.str("");
	ss << argv[3];
	ss >> numberOfSelectedDevice;

	ss.clear();
	ss.str("");
	ss << argv[4];
	ss >> ParticleEnergy;

	if ((selectedDeviceStart + numberOfSelectedDevice) < 0 || (selectedDeviceStart + numberOfSelectedDevice) > deviceCount) {
		std::cout << "You can only use the device from 0 to "<< deviceCount -1 <<std::endl;
		getchar();
		exit(1);
	}

	theError = hipSetDevice(selectedDeviceStart);
	if (theError != hipSuccess) {
		std::cout << "Set device "<< selectedDeviceStart << " error " << std::endl;
		getchar();
		exit(1);
	}


	/*Parameters*/
	NWSimParameters* theSimParemeters = new NWSimParameters();

	theSimParemeters->SetDefulatValue();

	NWBeam *theBeam = new NWBeam();

	theBeam->SetGunEnergy(ParticleEnergy*MeV);

	theSimParemeters->SetBeam(*theBeam);

	NWGlobal::GetInstance()->InitialGlobal(analysisMode,*theSimParemeters);

	NWInfoStore::GetInstance()->ReadEventsInfo(originalDataFilePath);

	std::cout << "Read done, execute the new analysis..." << std::endl;

	analysis.AnalysisResult(NWInfoStore::GetInstance()->GetEventsInfo());

	delete theSimParemeters;
	delete theBeam;
}

int main(int argc, char* argv[]) {

	AnalysisTheResult(argc, argv);

	return 0;
}
