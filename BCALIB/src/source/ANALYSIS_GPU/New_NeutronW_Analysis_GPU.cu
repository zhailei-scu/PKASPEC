#include <iostream>
#include "NWGlobal.h"
#include "NWAnalysis_GPU.cuh"
#include "NWSimParameters.h"
#include "hip/hip_runtime.h"
#include ""
#include <sstream>

void AnalysisTheResult(int argc, char* argv[]) {
	/*Local Vars*/
	stringstream ss;
	int deviceCount;
	int currentDevice;
	int selectedDeviceStart;
	int numberOfSelectedDevice;
	hipError_t theError;
	/*Body*/

	hipGetDeviceCount(&deviceCount);

	hipGetDevice(&currentDevice);

	NWAnalysis_GPU analysis;

	if (argc < 3) {
		std::cout << "The argument number is: " << argc << std::endl;
		std::cout << "You must special the 1) the original file path" << std::endl;
		std::cout << "You must special the 2) the start of selectted GPU ID" << std::endl;
		std::cout << "You must special the 3) the number of selectted GPU ID" << std::endl;
		system("pause");
		exit(1);
	}
	std::string originalDataFilePath;

	originalDataFilePath = std::string(argv[1]);

	ss.clear();
	ss.str("");
	ss << argv[2];
	ss >> selectedDeviceStart;

	ss.clear();
	ss.str("");
	ss << argv[3];
	ss >> numberOfSelectedDevice;

	if ((selectedDeviceStart + numberOfSelectedDevice) < 0 || (selectedDeviceStart + numberOfSelectedDevice) >= deviceCount) {
		std::cout << "You can only use the device from 0 to "<< numberOfSelectedDevice-1 <<std::endl;
		getchar();
		exit(1);
	}

	theError = hipSetDevice(selectedDeviceStart);
	if (theError != hipSuccess) {
		std::cout << "Set device "<< selectedDeviceStart << " error " << std::endl;
		getchar();
		exit(1);
	}

	NWGlobal::GetInstance()->InitialGlobal(analysisMode);

	NWInfoStore::GetInstance()->ReadEventsInfo(originalDataFilePath);

	std::cout << "Read done, execute the new analysis..." << std::endl;

	analysis.AnalysisResult(NWInfoStore::GetInstance()->GetEventsInfo());
}

int main(int argc, char* argv[]) {

	AnalysisTheResult(argc, argv);

	return 0;
}
