
#include "hip/hip_runtime.h"


__global__ void Kernel_Statistic();
__global__ void Kernel_Statistic() {
	int tid = threadIdx.y*blockDim.x + threadIdx.x;
	int bid = blockIdx.y*gridDim.x + blockIdx.x;
	int cid = bid * 1 + tid;




}

void Dev_Statistic() {
	Kernel_Statistic <<<1, 1 >>> ();
}