#include "hip/hip_runtime.h"
#include "NWAnalysis_GPU.cuh"
#include "NWGlobal.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <iomanip>
#include <algorithm>

__global__ void Kernel_Statistic(G4ThreeVector* Dev_ShiftPos,int *Dev_CeilZID, int *Dev_StartID,int *Dev_PKANumEachCeilZ,int *Dev_ResultInnerCeilZID, int *Dev_ResultCeilZID, int totalCeilZ,int NPKA) {
	int tid = threadIdx.y*blockDim.x + threadIdx.x;
	int bid = blockIdx.y*gridDim.x + blockIdx.x;
	int cid = bid * BLOCKSIZE + tid;
	int shellNum = 0;
	bool founded = false;
	G4ThreeVector subjectShiftPos;
	G4ThreeVector objectShiftPos;
	G4ThreeVector pKADist;
	double distance;
	double minDist;
	int resultInnerCeilZID;
	int resultCeilZID;
	/*Body*/

	minDist = 1.e32;

	if (cid < NPKA) {
		int CeilZID = Dev_CeilZID[cid];

		subjectShiftPos = Dev_ShiftPos[cid];

		while (!founded) {
			for (int z = std::max(CeilZID - shellNum,0); z < std::min(CeilZID + shellNum+1, totalCeilZ); z++) {

				for (int kk = Dev_StartID[z]; kk < Dev_StartID[z] + Dev_PKANumEachCeilZ[z]; kk++) {

					if (kk != cid) {
						objectShiftPos = Dev_ShiftPos[kk];

						pKADist = subjectShiftPos - objectShiftPos;
						distance = pKADist.mag();

						if (distance < minDist) {

							minDist = distance;

							founded = true;

							resultInnerCeilZID = kk;

							resultCeilZID = z;
						}


					}

				}

			}

			shellNum++;
		}

		Dev_ResultInnerCeilZID[cid] = resultInnerCeilZID;
		Dev_ResultCeilZID[cid] = resultCeilZID;
	}


}

void Dev_Statistic(std::vector<int>* linkedCells_EventID,
					std::vector<int>* linkedCells_TrackID,
					std::vector<int>* linkedCells_ZoneID,
					std::vector<StepInfo*>* linkedCells_StepInfo,
					std::vector<G4ThreeVector>* linkedCells_ShiftPos,
					int totalCeilZ, int totalSize,std::fstream* ofsAnalysisPath_DistanceXYZ) {

	/*Local Vars*/
	G4ThreeVector* ShiftPos;
	G4ThreeVector* Dev_ShiftPos;
	int *CeilZID;
	int *Dev_CeilZID;
	int *ResultInnerCeilZID;
	int *Dev_ResultInnerCeilZID;
	int *ResultCeilZID;
	int *Dev_ResultCeilZID;
	int *StartID;
	int *Dev_StartID;
	int *PKANumEachCeilZ;
	int *Dev_PKANumEachCeilZ;
	G4ThreeVector subjectTruePosition;
	G4ThreeVector objectTruePosition;
	G4ThreeVector subjectShifPos;
	G4ThreeVector objectShifPos;
	G4ThreeVector pKADist;
	hipError_t cudaStatus;
	int index;
	dim3 blocks;
	dim3 threads;
	int NB;
	int NBX;
	int NBY;
	int outwidth;
	/*Body*/

	outwidth = NWGlobal::GetInstance()->GetSimParamters().GetOutWidth();

	ShiftPos = new G4ThreeVector[totalSize];

	ResultInnerCeilZID = new int[totalSize];

	ResultCeilZID = new int[totalSize];

	CeilZID = new int[totalSize];

	StartID = new int[totalCeilZ];

	PKANumEachCeilZ = new int[totalCeilZ];
	index = 0;

	for (int i = 0; i < totalCeilZ; i++) {

		StartID[i] = index;

		for (std::vector<G4ThreeVector>::iterator it = linkedCells_ShiftPos[i].begin; it != linkedCells_ShiftPos[i].end; it++) {
			ShiftPos[index] = *it;
			CeilZID[index] = i;

			index++;
		}

		PKANumEachCeilZ[i] = linkedCells_ShiftPos[i].size();
	}

	cudaStatus = hipMalloc((void**)&Dev_ShiftPos, sizeof(G4ThreeVector)*totalSize);
	if (cudaStatus != hipSuccess) {
		std::cout << "The memory allocate not right !" << std::endl;
		system("pause");
		exit(1);
	}

	cudaStatus = hipMalloc((void**)&Dev_ResultInnerCeilZID, sizeof(int)*totalSize);
	if (cudaStatus != hipSuccess) {
		std::cout << "The memory allocate not right !" << std::endl;
		system("pause");
		exit(1);
	}

	cudaStatus = hipMalloc((void**)&Dev_ResultCeilZID, sizeof(int)*totalSize);
	if (cudaStatus != hipSuccess) {
		std::cout << "The memory allocate not right !" << std::endl;
		system("pause");
		exit(1);
	}

	cudaStatus = hipMalloc((void**)&Dev_CeilZID, sizeof(int)*totalSize);
	if (cudaStatus != hipSuccess) {
		std::cout << "The memory allocate not right !" << std::endl;
		system("pause");
		exit(1);
	}


	cudaStatus = hipMalloc((void**)&Dev_StartID, sizeof(int)*totalCeilZ);
	if (cudaStatus != hipSuccess) {
		std::cout << "The memory allocate not right !" << std::endl;
		system("pause");
		exit(1);
	}

	cudaStatus = hipMalloc((void**)&Dev_PKANumEachCeilZ, sizeof(int)*totalCeilZ);
	if (cudaStatus != hipSuccess) {
		std::cout << "The memory allocate not right !" << std::endl;
		system("pause");
		exit(1);
	}

	cudaStatus = hipMemcpy(Dev_ShiftPos, ShiftPos, sizeof(G4ThreeVector)*totalSize, hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(Dev_CeilZID, CeilZID, sizeof(int)*totalSize, hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(Dev_StartID, StartID, sizeof(int)*totalCeilZ, hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(Dev_PKANumEachCeilZ, PKANumEachCeilZ, sizeof(int)*totalCeilZ, hipMemcpyHostToDevice);

	NB = (totalSize - 1) / BLOCKSIZE + 1;

	NBX = GRIDDIMX;
	NBY = (NB - 1) / NBX + 1;

	blocks = dim3(NBX, NBY);
	threads = dim3(BLOCKSIZE, 1);

	Kernel_Statistic<<<blocks, threads >>>(Dev_ShiftPos, Dev_CeilZID, Dev_StartID, Dev_PKANumEachCeilZ,Dev_ResultInnerCeilZID, Dev_ResultCeilZID,totalCeilZ, totalSize);

	cudaStatus = hipMemcpy(ResultInnerCeilZID, Dev_ResultInnerCeilZID, sizeof(int)*totalSize, hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(ResultCeilZID, Dev_ResultCeilZID, sizeof(int)*totalSize, hipMemcpyDeviceToHost);
	/*OutPut*/

	index = 0;

	for (int i = 0; i < totalCeilZ; i++) {

		for (int j = 0; j<linkedCells_ShiftPos[i].size(); j++) {
			subjectShifPos = linkedCells_ShiftPos[i].at(j);
			objectShifPos = linkedCells_ShiftPos[ResultCeilZID[index]].at(ResultInnerCeilZID[index]);
			pKADist = subjectShifPos - objectShifPos;

			if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {
				subjectTruePosition = linkedCells_StepInfo[i].at(j)->GetpostPosition();
				objectTruePosition = linkedCells_StepInfo[ResultCeilZID[index]].at(ResultInnerCeilZID[index])->GetpostPosition();
			}
			else if (ConcentReaction(InletToFirstNonEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
				ConcentReaction(InletEstAndInEstTillEnd) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
				ConcentReaction(MatrixAtom) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
				ConcentReaction(Iso) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

				subjectTruePosition = linkedCells_StepInfo[i].at(j)->GetprePosition();
				objectTruePosition = linkedCells_StepInfo[ResultCeilZID[index]].at(ResultInnerCeilZID[index])->GetprePosition();
			}


			*ofsAnalysisPath_DistanceXYZ
				<< std::setw(outwidth) << linkedCells_ZoneID[i].at(j)
				<< std::setw(outwidth) << i
				<< std::setw(outwidth) << linkedCells_EventID[i].at(j)
				<< std::setw(outwidth) << linkedCells_TrackID[i].at(j)
				<< std::setw(outwidth) << linkedCells_StepInfo[i].at(i)->GetStepID()
				<< std::setw(outwidth) << linkedCells_ZoneID[ResultCeilZID[index]].at[ResultInnerCeilZID[index]]
				<< std::setw(outwidth) << j
				<< std::setw(outwidth) << linkedCells_EventID[ResultCeilZID[index]].at[ResultInnerCeilZID[index]]
				<< std::setw(outwidth) << linkedCells_TrackID[ResultCeilZID[index]].at[ResultInnerCeilZID[index]]
				<< std::setw(outwidth) << linkedCells_StepInfo[ResultCeilZID[index]].at[ResultInnerCeilZID[index]]->GetStepID()
				<< std::setw(outwidth) << subjectTruePosition.getX()
				<< std::setw(outwidth) << subjectTruePosition.getY()
				<< std::setw(outwidth) << subjectTruePosition.getZ()
				<< std::setw(outwidth) << subjectShifPos.getX()
				<< std::setw(outwidth) << subjectShifPos.getY()
				<< std::setw(outwidth) << subjectShifPos.getZ()
				<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << abs(pKADist.getX())
				<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << abs(pKADist.getY())
				<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << abs(pKADist.getZ())
				<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << pKADist.mag() << std::endl;


			index++;
		}
	}


	/*Memory Free*/
	if (NULL != ShiftPos) delete[] ShiftPos;
	if (NULL != ShiftPos) delete[] ResultInnerCeilZID;
	if (NULL != ShiftPos) delete[] ResultCeilZID;
	if (NULL != ShiftPos) delete[] StartID;
	if (NULL != ShiftPos) delete[] PKANumEachCeilZ;

	hipFree(Dev_ShiftPos);
	hipFree(Dev_CeilZID);
	hipFree(Dev_ResultInnerCeilZID);
	hipFree(Dev_ResultCeilZID);
	hipFree(Dev_StartID);
	hipFree(Dev_PKANumEachCeilZ);

}


void Dev_Cal_MinDist_LinkedCell(std::map<int, std::vector<TrackInfo>>* storedData, double boundary[][2],
	fstream* ofsAnalysisPath_DistanceXYZ, fstream *ofsAnalysisPath_linkedCellPosition,
	fstream* ofsAnalysisPath_ZoneCount, fstream* ofsAnalysisPath_CeilCount) {
	/*Local Vars*/
	double ceil_Interval[3];
	int ceilingNum_OneDim[3];
	int ceilIndex[3];
	double newBoundary[3][2];
	double beamCenter[2];
	int linkID;
	int subjectLinkID;
	int objectLinkID;
	std::map<int, std::vector<TrackInfo>>::iterator it;
	std::vector<TrackInfo>::iterator iteratorTrackInfo;
	std::vector<int>* linkedCells_EventID = NULL;
	std::vector<int>* linkedCells_TrackID = NULL;
	std::vector<int>* linkedCells_ZoneID = NULL;
	std::vector<StepInfo*>* linkedCells_StepInfo = NULL;
	std::vector<G4ThreeVector>* linkedCells_ShiftPos = NULL;
	int** ceilXYCount;
	G4ThreeVector shiftPosition;
	G4ThreeVector subjectTruePosition;
	G4ThreeVector objectTruePosition;
	G4ThreeVector subjectShiftPostion;
	G4ThreeVector objectShiftPostion;
	G4ThreeVector pKADist;
	double distance;
	int outwidth;
	int ZoneNum;
	int ZoneCenter[2];
	int SubjectZoneID;
	int SubjectZoneID_X;
	int SubjectZoneID_Y;
	int ObjectZoneID;
	int ObjectZoneID_X;
	int ObjectZoneID_Y;
	int theSize;
	int subjectEventID;
	int subjectTrackID;
	int subjectStepID;
	double minDist;
	int minDist_ObjectLinkedID;
	int minDist_objectEventID;
	int minDist_objectTrackID;
	int minDist_objectStepID;
	double minDist_x;
	double minDist_y;
	double minDist_z;
	int shellNum;
	bool founded;
	int iInterval;
	int ZoneCount;
	int corr_i;
	int corr_j;
	int totalSize;
	/*Body*/

	ceil_Interval[0] = ceil_Interval[1] = NWGlobal::GetInstance()->GetSimParamters().GetLinkCellInterval_xy();

	ceil_Interval[2] = NWGlobal::GetInstance()->GetSimParamters().GetLinkCellInterval_z();

	NWGlobal::GetInstance()->GetSimParamters().GetNWBeam().GetFluxCenter(beamCenter);

	for (int i = 0; i <= 1; i++) {
		ceilingNum_OneDim[i] = 2 * ceil(max(beamCenter[i] - boundary[i][0], boundary[i][1] - beamCenter[i]) / ceil_Interval[i]) + 1;
		ceilingNum_OneDim[i] = max(ceilingNum_OneDim[i], 3);
	}

	ceilingNum_OneDim[0] = ceilingNum_OneDim[1] = max(ceilingNum_OneDim[0], ceilingNum_OneDim[1]);

	ceilingNum_OneDim[2] = ceil((boundary[2][1] - boundary[2][0]) / ceil_Interval[2]);
	ceilingNum_OneDim[2] = max(ceilingNum_OneDim[2], 1);

	for (int i = 0; i < 3; i++) {
		ceilIndex[i] = 0;
	}

	std::cout << "boundary_x " << boundary[0][0] << " " << boundary[0][1] << std::endl;
	std::cout << "boundary_y " << boundary[1][0] << " " << boundary[1][1] << std::endl;
	std::cout << "boundary_z " << boundary[2][0] << " " << boundary[2][1] << std::endl;

	std::cout << "cellnum_x " << ceilingNum_OneDim[0] << std::endl;
	std::cout << "cellnum_y " << ceilingNum_OneDim[1] << std::endl;
	std::cout << "cellnum_z " << ceilingNum_OneDim[2] << std::endl;


	/*Zone ID start from 0(Center zone)*/
	ZoneNum = ceilingNum_OneDim[0] / 2 + 1;
	ZoneCenter[0] = ZoneNum - 1;
	ZoneCenter[1] = ZoneCenter[0];

	outwidth = NWGlobal::GetInstance()->GetSimParamters().GetOutWidth();


	for (int i = 0; i <= 1; i++) {
		newBoundary[i][0] = (beamCenter[i] - (ZoneNum - 1 + 0.50)*ceil_Interval[i]);
		newBoundary[i][1] = (beamCenter[i] + (ZoneNum - 1 + 0.50)*ceil_Interval[i]);
	}
	newBoundary[2][0] = boundary[2][0];
	newBoundary[2][1] = boundary[2][0] + ceilingNum_OneDim[2] * ceil_Interval[2];


	std::cout << "new_boundary_x " << newBoundary[0][0] << " " << newBoundary[0][1] << std::endl;
	std::cout << "new_boundary_y " << newBoundary[1][0] << " " << newBoundary[1][1] << std::endl;
	std::cout << "new_boundary_z " << newBoundary[2][0] << " " << newBoundary[2][1] << std::endl;


	linkedCells_EventID = new std::vector<int>[ceilingNum_OneDim[2]];
	linkedCells_TrackID = new std::vector<int>[ceilingNum_OneDim[2]];
	linkedCells_ZoneID = new std::vector<int>[ceilingNum_OneDim[2]];
	linkedCells_StepInfo = new std::vector<StepInfo*>[ceilingNum_OneDim[2]];
	linkedCells_ShiftPos = new std::vector<G4ThreeVector>[ceilingNum_OneDim[2]];


	ceilXYCount = new int*[ceilingNum_OneDim[0]];
	for (int i = 0; i < ceilingNum_OneDim[0]; i++) {
		ceilXYCount[i] = new int[ceilingNum_OneDim[1]];

		for (int j = 0; j < ceilingNum_OneDim[1]; j++) {
			ceilXYCount[i][j] = 0;
		}
	}


	for (int i = 0; i < ceilingNum_OneDim[2]; i++) {
		std::vector<int>().swap(linkedCells_EventID[i]);
		std::vector<int>().swap(linkedCells_TrackID[i]);
		std::vector<int>().swap(linkedCells_ZoneID[i]);
		std::vector<StepInfo*>().swap(linkedCells_StepInfo[i]);
	}


	for (int k = 0; k < ceilingNum_OneDim[2]; k++) {
		for (int j = 0; j < ceilingNum_OneDim[1]; j++) {
			SubjectZoneID_Y = max(ZoneCenter[1] - j, j - ZoneCenter[1]);

			for (int i = 0; i < ceilingNum_OneDim[0]; i++) {

				SubjectZoneID_X = max(ZoneCenter[0] - i, i - ZoneCenter[0]);

				SubjectZoneID = max(SubjectZoneID_X, SubjectZoneID_Y);

				*ofsAnalysisPath_linkedCellPosition
					<< std::setw(outwidth) << k * ceilingNum_OneDim[0] * ceilingNum_OneDim[1] + j * ceilingNum_OneDim[0] + i
					<< std::setw(outwidth) << SubjectZoneID
					<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << (i - ZoneCenter[0])*ceil_Interval[0]
					<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << (j - ZoneCenter[1])*ceil_Interval[1]
					<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << newBoundary[2][0] + k * ceil_Interval[2]
					<< std::endl;
			}
		}
	}


	totalSize = 0;
	it = storedData->begin();

	for (; it != storedData->end(); it++) {
		iteratorTrackInfo = it->second.begin();

		for (; iteratorTrackInfo != it->second.end(); iteratorTrackInfo++) {

			theSize = iteratorTrackInfo->GetStepsInfo()->size();

			for (int index = 0; index < theSize; index++) {

				totalSize++;

				if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {
					if (0 != iteratorTrackInfo->GetStepsInfo()->at(index).GetProcessName().compare(std::string("hadElastic"))) {
						break;
					}
				}

				if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

					if (iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getX() < newBoundary[0][0] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getX() > newBoundary[0][1] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getY() < newBoundary[1][0] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getY() > newBoundary[1][1] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getZ() < newBoundary[2][0] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getZ() > newBoundary[2][1]) {
						continue;
					}

					ceilIndex[0] = min(int((iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getX() - newBoundary[0][0]) / ceil_Interval[0]), ceilingNum_OneDim[0] - 1);
					ceilIndex[1] = min(int((iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getY() - newBoundary[1][0]) / ceil_Interval[1]), ceilingNum_OneDim[1] - 1);
					ceilIndex[2] = min(int((iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition().getZ() - newBoundary[2][0]) / ceil_Interval[2]), ceilingNum_OneDim[2] - 1);

					shiftPosition = iteratorTrackInfo->GetStepsInfo()->at(index).GetpostPosition();

				}
				else if (ConcentReaction(InletToFirstNonEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
					ConcentReaction(InletEstAndInEstTillEnd) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
					ConcentReaction(MatrixAtom) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
					ConcentReaction(Iso) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {


					if (iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getX() < newBoundary[0][0] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getX() > newBoundary[0][1] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getY() < newBoundary[1][0] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getY() > newBoundary[1][1] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getZ() < newBoundary[2][0] ||
						iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getZ() > newBoundary[2][1]) {

						continue;
					}

					ceilIndex[0] = min(int((iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getX() - newBoundary[0][0]) / ceil_Interval[0]), ceilingNum_OneDim[0] - 1);
					ceilIndex[1] = min(int((iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getY() - newBoundary[1][0]) / ceil_Interval[1]), ceilingNum_OneDim[1] - 1);
					ceilIndex[2] = min(int((iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition().getZ() - newBoundary[2][0]) / ceil_Interval[2]), ceilingNum_OneDim[2] - 1);

					shiftPosition = iteratorTrackInfo->GetStepsInfo()->at(index).GetprePosition();
				}

				ceilIndex[0] = max(ceilIndex[0], 0);
				ceilIndex[1] = max(ceilIndex[1], 0);
				ceilIndex[2] = max(ceilIndex[2], 0);

				SubjectZoneID_X = max(ZoneCenter[0] - ceilIndex[0], ceilIndex[0] - ZoneCenter[0]);
				SubjectZoneID_Y = max(ZoneCenter[1] - ceilIndex[1], ceilIndex[1] - ZoneCenter[1]);
				SubjectZoneID = max(SubjectZoneID_X, SubjectZoneID_Y);


				ceilXYCount[ceilIndex[0]][ceilIndex[1]] += 1;

				shiftPosition[0] = shiftPosition[0] - (ceilIndex[0] - ZoneCenter[0])*ceil_Interval[0];
				shiftPosition[1] = shiftPosition[1] - (ceilIndex[1] - ZoneCenter[1])*ceil_Interval[1];

				linkID = ceilIndex[2];
				linkedCells_ZoneID[linkID].push_back(SubjectZoneID);
				linkedCells_EventID[linkID].push_back(it->first);
				linkedCells_TrackID[linkID].push_back(iteratorTrackInfo->GetTrackID());
				linkedCells_StepInfo[linkID].push_back(&iteratorTrackInfo->GetStepsInfo()->at(index));
				linkedCells_ShiftPos[linkID].push_back(shiftPosition);
			}

		}
	}


	Dev_Statistic(linkedCells_EventID,
					linkedCells_TrackID,
					linkedCells_ZoneID,
					linkedCells_StepInfo,
					linkedCells_ShiftPos,
					ceilingNum_OneDim[2], totalSize,ofsAnalysisPath_DistanceXYZ);


	for (int ZoneID = 0; ZoneID < ZoneNum; ZoneID++) {

		ZoneCount = 0;

		for (int j = -ZoneID; j <= ZoneID; j = j + 1) {

			if (j == -ZoneID || j == ZoneID) {
				iInterval = 1;
			}
			else {
				iInterval = 2 * ZoneID;
			}

			corr_j = ZoneCenter[1] + j;


			for (int i = -ZoneID; i <= ZoneID; i = i + iInterval) {

				corr_i = ZoneCenter[0] + i;

				ZoneCount += ceilXYCount[corr_i][corr_j];

				*ofsAnalysisPath_CeilCount
					<< std::setw(outwidth) << ZoneID
					<< std::setw(outwidth) << corr_j * ceilingNum_OneDim[0] + corr_i
					<< std::setw(outwidth) << i
					<< std::setw(outwidth) << j
					<< std::setw(outwidth) << ceilXYCount[corr_i][corr_j] << std::endl;

			}
		}


		*ofsAnalysisPath_ZoneCount
			<< std::setw(outwidth) << ZoneID
			<< std::setw(outwidth) << ZoneCount << std::endl;

	}



	if (NULL != linkedCells_EventID) delete[] linkedCells_EventID;
	if (NULL != linkedCells_TrackID) delete[] linkedCells_TrackID;
	if (NULL != linkedCells_ZoneID) delete[] linkedCells_ZoneID;
	if (NULL != linkedCells_StepInfo) delete[] linkedCells_StepInfo;

	for (int i = 0; i < ceilingNum_OneDim[0]; i++) {
		delete[] ceilXYCount[i];
		ceilXYCount[i] = NULL;
	}
	delete[] ceilXYCount;

}


NWAnalysis_GPU::NWAnalysis_GPU() {

}

NWAnalysis_GPU::~NWAnalysis_GPU() {
}

void NWAnalysis_GPU::AnalysisResult(std::map<int, std::vector<TrackInfo>>* storedData) {
	std::fstream ofsOrignalDistance;
	std::fstream ofsAnalysis_EqualInterval;
	std::fstream ofsAnalysis_PowerInterval;
	std::fstream ofsAnalysis_EndReason;
	std::fstream ofsAnalysis_DeviateAxesDistance;
	std::fstream ofsAnalysisPath_DistanceXYZ;
	std::fstream ofsAnalysisPath_linkedCellPosition;
	std::fstream ofsAnalysisPath_ZoneCount;
	std::fstream ofsAnalysisPath_CeilXYCount;
	std::string OrignalDistancePath;
	std::string AnalysisPath_EqualInterval;
	std::string AnalysisPath_PowerInterval;
	std::string AnalysisPath_EndReason;
	std::string AnalysisPath_DeviateAxesDistance;
	std::string AnalysisPath_DistanceXYZ;
	std::string AnalysisPath_linkedCellPosition;
	std::string AnalysisPath_ZoneCount;
	std::string AnalysisPath_CeilXYCount;
	std::vector<double> theDistance;
	double maxDistance = -1.0;
	double minDistance = 1E32;
	int minLog;
	int maxLog;
	int binNum;
	double *binEnds;
	double *score;
	std::stringstream ss;
	int PowerInterval_BinNum = 200;
	double PowerInterval_Min = 0.01;
	double PowerInterval_Max = 200;
	double PowerInterval_DeltaLength;
	double *binEnds_PowerInterval;
	int *score_PowerInterval;
	int outwidth;
	std::map<int, std::vector<TrackInfo>>::iterator it;
	std::vector<TrackInfo>::iterator iteratorTrackInfo;
	std::vector<StepInfo>::iterator iteratorStepInfo;
	G4ThreeVector prePKAPos;
	G4ThreeVector postPKAPos;
	int index;
	G4ThreeVector pKADist;
	double distance;
	double distanceToOriginAxe;
	G4ThreeVector vectorToOrigin;
	double vectorMultiple;
	double magOriginDirection;
	double magvectorToOrigin;
	double boundary[3][2];
	//---Body---

	boundary[0][0] = 1.e32;
	boundary[1][0] = 1.e32;
	boundary[2][0] = 1.e32;
	boundary[0][1] = -1.e32;
	boundary[1][1] = -1.e32;
	boundary[2][1] = -1.e32;

	outwidth = NWGlobal::GetInstance()->GetSimParamters().GetOutWidth();

	binEnds_PowerInterval = new double[PowerInterval_BinNum];

	score_PowerInterval = new int[PowerInterval_BinNum];

	PowerInterval_DeltaLength = (std::log10(PowerInterval_Max) - std::log10(PowerInterval_Min)) / PowerInterval_BinNum;

	score = NULL;
	binEnds = NULL;


	if (NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->length() > 0) {

		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_OriginDistance.txt";

		ss >> OrignalDistancePath;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_Analysis_EqualInterval.txt";

		ss >> AnalysisPath_EqualInterval;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_Analysis_PowerInterval.txt";

		ss >> AnalysisPath_PowerInterval;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_Analysis_EndReason.txt";

		ss >> AnalysisPath_EndReason;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_Analysis_DeviateAxesDistance.txt";

		ss >> AnalysisPath_DeviateAxesDistance;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_Analysis_DistanceXYZ.txt";

		ss >> AnalysisPath_DistanceXYZ;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_DistanceResult_Analysis_linkedCellPosition.txt";

		ss >> AnalysisPath_linkedCellPosition;


		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_ZoneCount.txt";

		ss >> AnalysisPath_ZoneCount;



		ss.clear();

		ss.str("");

		ss << NWGlobal::GetInstance()->GetSimParamters().GetOutPath()->c_str() << "\\" << "New_CeilCount.txt";

		ss >> AnalysisPath_CeilXYCount;

	}
	else {
		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_OriginDistance.txt";

		ss >> OrignalDistancePath;


		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_Analysis_EqualInterval.txt";

		ss >> AnalysisPath_EqualInterval;


		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_Analysis_PowerInterval.txt";

		ss >> AnalysisPath_PowerInterval;


		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_Analysis_EndReason.txt";

		ss >> AnalysisPath_EndReason;


		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_Analysis_DeviateAxesDistance.txt";

		ss >> AnalysisPath_DeviateAxesDistance;


		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_Analysis_DistanceXYZ.txt";

		ss >> AnalysisPath_DistanceXYZ;


		ss.clear();

		ss.str("");

		ss << "New_DistanceResult_Analysis_linkedCellPosition.txt";

		ss >> AnalysisPath_linkedCellPosition;



		ss.clear();

		ss.str("");

		ss << "New_ZoneCount.txt";

		ss >> AnalysisPath_ZoneCount;


		ss.clear();

		ss.str("");

		ss << "New_CeilCount.txt";

		ss >> AnalysisPath_CeilXYCount;

	}

	ofsOrignalDistance.open(OrignalDistancePath, std::ios::out | std::ios::ate);

	ofsAnalysis_EqualInterval.open(AnalysisPath_EqualInterval, std::ios::out | std::ios::ate);

	ofsAnalysis_PowerInterval.open(AnalysisPath_PowerInterval, std::ios::out | std::ios::ate);

	ofsAnalysis_EndReason.open(AnalysisPath_EndReason, std::ios::out | std::ios::ate);

	ofsAnalysis_EndReason << std::setw(outwidth) << "EventID:"
		<< std::setw(outwidth) << "EndReason" << std::setw(outwidth) << "EndEnergy" << std::endl;

	ofsAnalysis_DeviateAxesDistance.open(AnalysisPath_DeviateAxesDistance, std::ios::out | std::ios::ate);

	ofsAnalysis_DeviateAxesDistance << std::setw(outwidth) << "EventID:"
		<< std::setw(outwidth) << "ToOrgVector(mm)" << std::endl;


	ofsAnalysisPath_DistanceXYZ.open(AnalysisPath_DistanceXYZ, std::ios::out | std::ios::ate);

	ofsAnalysisPath_DistanceXYZ
		<< std::setw(outwidth) << "SubjectZONEID"
		<< std::setw(outwidth) << "SubjectLinkedID"
		<< std::setw(outwidth) << "SubjectEventID"
		<< std::setw(outwidth) << "SubjectTrackID"
		<< std::setw(outwidth) << "SubjectStepID"
		<< std::setw(outwidth) << "ObjectZONEID"
		<< std::setw(outwidth) << "ObjectLinkedID"
		<< std::setw(outwidth) << "ObjectEventID"
		<< std::setw(outwidth) << "ObjectTrackID"
		<< std::setw(outwidth) << "ObjectStepID"
		<< std::setw(outwidth) << "TruePos_X(mm)"
		<< std::setw(outwidth) << "TruePos_Y(mm)"
		<< std::setw(outwidth) << "TruePos_Z(mm)"
		<< std::setw(outwidth) << "ShiftPos_X(mm)"
		<< std::setw(outwidth) << "ShiftPos_Y(mm)"
		<< std::setw(outwidth) << "ShiftPos_Z(mm)"
		<< std::setw(outwidth) << "MinDeltaX(mm)"
		<< std::setw(outwidth) << "MinDeltaY(mm)"
		<< std::setw(outwidth) << "MinDeltaZ(mm)"
		<< std::setw(outwidth) << "MinDeltaDist(mm)" << std::endl;


	ofsAnalysisPath_linkedCellPosition.open(AnalysisPath_linkedCellPosition, std::ios::out | std::ios::ate);

	ofsAnalysisPath_linkedCellPosition
		<< std::setw(outwidth) << "CellID"
		<< std::setw(outwidth) << "ZoneID"
		<< std::setw(outwidth) << "cent_x(mm)"
		<< std::setw(outwidth) << "cent_y(mm)"
		<< std::setw(outwidth) << "cent_z(mm)" << std::endl;


	ofsAnalysisPath_ZoneCount.open(AnalysisPath_ZoneCount, std::ios::out | std::ios::ate);

	ofsAnalysisPath_ZoneCount
		<< std::setw(outwidth) << "ZoneID"
		<< std::setw(outwidth) << "Count" << std::endl;



	ofsAnalysisPath_CeilXYCount.open(AnalysisPath_CeilXYCount, std::ios::out | std::ios::ate);

	ofsAnalysisPath_CeilXYCount
		<< std::setw(outwidth) << "ZoneID"
		<< std::setw(outwidth) << "CeilXYID"
		<< std::setw(outwidth) << "ZoneX"
		<< std::setw(outwidth) << "ZoneY"
		<< std::setw(outwidth) << "Count" << std::endl;


	it = storedData->begin();

	for (; it != storedData->end(); it++) {
		iteratorTrackInfo = it->second.begin();

		index = 0;

		for (; iteratorTrackInfo != it->second.end(); iteratorTrackInfo++) {

			iteratorStepInfo = iteratorTrackInfo->GetStepsInfo()->begin();

			if (iteratorTrackInfo->GetStepsInfo()->size() > 0) {

				for (; iteratorStepInfo != iteratorTrackInfo->GetStepsInfo()->end(); iteratorStepInfo++) {

					if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

						if (0 != iteratorStepInfo->GetProcessName().compare(std::string("hadElastic"))) {

							ofsAnalysis_EndReason << std::setw(outwidth) << it->first
								<< std::setw(outwidth) << StepInfo::ConvertToProcessID(iteratorStepInfo->GetProcessName())
								<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << iteratorStepInfo->GetpreEng()
								<< std::endl;
							break;
						}

					}

					index++;

					if (index <= 1) {

						if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {
							prePKAPos = iteratorStepInfo->GetpostPosition();
							postPKAPos = iteratorStepInfo->GetpostPosition();

						}
						else if (ConcentReaction(InletToFirstNonEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
							ConcentReaction(InletEstAndInEstTillEnd) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
							ConcentReaction(MatrixAtom) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
							ConcentReaction(Iso) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

							prePKAPos = iteratorStepInfo->GetprePosition();
							postPKAPos = iteratorStepInfo->GetprePosition();
						}
					}
					else {

						if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {
							postPKAPos = iteratorStepInfo->GetpostPosition();

						}
						else if (ConcentReaction(InletToFirstNonEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
							ConcentReaction(InletEstAndInEstTillEnd) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
							ConcentReaction(MatrixAtom) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
							ConcentReaction(Iso) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

							postPKAPos = iteratorStepInfo->GetprePosition();
						}

						pKADist = prePKAPos - postPKAPos;

						distance = pKADist.mag();

						ofsOrignalDistance << std::setiosflags(std::ios::scientific) << std::setprecision(7) << distance << std::endl;

						prePKAPos = postPKAPos;

						theDistance.push_back(distance);

						maxDistance = std::max(maxDistance, distance);
						minDistance = std::min(minDistance, distance);
					}

					if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {
						vectorToOrigin = iteratorStepInfo->GetpostPosition() - iteratorStepInfo->GetOriginPosition();

						boundary[0][0] = min(iteratorStepInfo->GetpostPosition().getX(), boundary[0][0]);
						boundary[1][0] = min(iteratorStepInfo->GetpostPosition().getY(), boundary[1][0]);
						boundary[2][0] = min(iteratorStepInfo->GetpostPosition().getZ(), boundary[2][0]);
						boundary[0][1] = max(iteratorStepInfo->GetpostPosition().getX(), boundary[0][1]);
						boundary[1][1] = max(iteratorStepInfo->GetpostPosition().getY(), boundary[1][1]);
						boundary[2][1] = max(iteratorStepInfo->GetpostPosition().getZ(), boundary[2][1]);

					}
					else if (ConcentReaction(InletToFirstNonEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
						ConcentReaction(InletEstAndInEstTillEnd) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
						ConcentReaction(MatrixAtom) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
						ConcentReaction(Iso) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

						vectorToOrigin = iteratorStepInfo->GetprePosition() - iteratorStepInfo->GetOriginPosition();

						boundary[0][0] = min(iteratorStepInfo->GetprePosition().getX(), boundary[0][0]);
						boundary[1][0] = min(iteratorStepInfo->GetprePosition().getY(), boundary[1][0]);
						boundary[2][0] = min(iteratorStepInfo->GetprePosition().getZ(), boundary[2][0]);
						boundary[0][1] = max(iteratorStepInfo->GetprePosition().getX(), boundary[0][1]);
						boundary[1][1] = max(iteratorStepInfo->GetprePosition().getY(), boundary[1][1]);
						boundary[2][1] = max(iteratorStepInfo->GetprePosition().getZ(), boundary[2][1]);

					}

					vectorMultiple = vectorToOrigin * iteratorStepInfo->GetOriginDirection();

					magvectorToOrigin = vectorToOrigin.mag();

					magOriginDirection = iteratorStepInfo->GetOriginDirection().mag();

					if (magvectorToOrigin <= 0 || magOriginDirection <= 0) {
						distanceToOriginAxe = 0.0;
					}
					else {
						distanceToOriginAxe = magvectorToOrigin * std::sqrt(1 - pow(std::fabs(vectorMultiple) / (magvectorToOrigin*magOriginDirection), 2));
					}

					ofsAnalysis_DeviateAxesDistance << std::setw(outwidth) << it->first
						<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << distanceToOriginAxe
						<< std::endl;

				}
			}

		}
	}

	if (ConcentReaction(InletToLastEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
		ConcentReaction(InletToFirstNonEst) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
		ConcentReaction(InletEstAndInEstTillEnd) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction() ||
		ConcentReaction(MatrixAtom) == NWGlobal::GetInstance()->GetSimParamters().GetTheConcentReaction()) {

		std::cout << "minDistance: " << minDistance << std::endl;
		std::cout << "maxDistance: " << maxDistance << std::endl;

		minDistance = std::max(minPrecision, minDistance);

		minLog = std::floor(std::log10(minDistance));
		maxLog = std::ceil(std::log10(maxDistance));

		binNum = (maxLog - minLog)*BinNumberEachPower;

		binEnds = new double[binNum];
		score = new double[binNum];

		for (int i = 0; i < binNum; i++) {
			double base = pow(10, minLog + i / BinNumberEachPower);
			binEnds[i] = base + (i % BinNumberEachPower)*base * 9 / BinNumberEachPower;
			score[i] = 0;
		}

		for (int i = 0; i < PowerInterval_BinNum; i++) {
			binEnds_PowerInterval[i] = PowerInterval_Min * std::pow(10, i*PowerInterval_DeltaLength);
			score_PowerInterval[i] = 0;
		}

		for (std::vector<double>::iterator itDistance = theDistance.begin(); itDistance != theDistance.end(); itDistance++) {

			/*
			int  LogPos = std::floor(std::log10(*itDistance));

			int truePos = (LogPos - minLog)*NWGlobal::GetInstance()->BinNumberEachPower
				+ floor(NWGlobal::GetInstance()->BinNumberEachPower*((*itDistance) / pow(10, LogPos)) / 9);

			score[truePos]++;
			*/

			for (int i = 0; i < binNum; i++) {

				if (*itDistance < binEnds[i]) {
					score[i]++;
					break;
				}
			}

			if (*itDistance >= PowerInterval_Min && *itDistance <= PowerInterval_Max) {
				int binPos = std::floor((std::log10(*itDistance) - std::log10(PowerInterval_Min)) / PowerInterval_DeltaLength);
				score_PowerInterval[binPos]++;
			}

		}


		for (int i = 0; i < binNum; i++) {

			score[i] = score[i] / pow(10, std::max(i - 1, 0) / BinNumberEachPower);

		}


		for (int i = 0; i < binNum; i++) {
			ofsAnalysis_EqualInterval << std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << binEnds[i]
				<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << score[i] << std::endl;
		}


		for (int i = 0; i < PowerInterval_BinNum; i++) {
			ofsAnalysis_PowerInterval << std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << binEnds_PowerInterval[i]
				<< std::setw(outwidth) << std::setiosflags(std::ios::scientific) << std::setprecision(7) << score_PowerInterval[i] << std::endl;
		}

	}

	Dev_Cal_MinDist_LinkedCell(storedData, boundary, &ofsAnalysisPath_DistanceXYZ, &ofsAnalysisPath_linkedCellPosition, &ofsAnalysisPath_ZoneCount, &ofsAnalysisPath_CeilXYCount);


	ofsAnalysis_EqualInterval.close();

	ofsAnalysis_PowerInterval.close();

	ofsOrignalDistance.close();

	ofsAnalysis_EndReason.close();

	ofsAnalysis_DeviateAxesDistance.close();

	ofsAnalysisPath_DistanceXYZ.close();

	ofsAnalysisPath_linkedCellPosition.close();

	ofsAnalysisPath_ZoneCount.close();

	ofsAnalysisPath_CeilXYCount.close();

	if (NULL != binEnds) delete[] binEnds;
	if (NULL != score) delete[] score;

	if (NULL != binEnds_PowerInterval) delete[] binEnds_PowerInterval;
	if (NULL != score_PowerInterval) delete[] score_PowerInterval;
}